#include <cstdlib>
#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>
#include "heap_manager.hpp"
#include "heap_allocator.hpp"
#include "utility.hpp"
using namespace std;

//#define USE_CUDA_ALLOCATOR
#undef USE_CUDA_ALLOCATOR
namespace ckt {
  HeapManager gHeapManager;
  int HeapManager::max_device_ids = 32;

  HeapManager::HeapManager() {
    m_gpu_heap_allocator.resize(HeapManager::max_device_ids, nullptr);
  }

  HeapManager::~HeapManager()
  {
    for (auto i = m_gpu_heap_allocator.begin(); i != m_gpu_heap_allocator.end(); i++) {
      if (*i != nullptr)
        delete *i;
    }
  }

  void *HeapManager::Malloc(Memory_Type type, const size_t &size)
  {
    void *addr; 
    if (type == CPU_HEAP)
      {
        addr = (void *)malloc(size);
#ifdef _DEBUG
        mCpuMemoryTracker.insert( pair<void *, int>(*addr, size));
        curCpuUsage += size;
        maxCpuUsage = maxCpuUsage > curCpuUsage? maxCpuUsage: curCpuUsage;
#endif      
      }
    else if (type == GPU_HEAP)
      {
        // init gpu allocator if not exist
#ifdef USE_CUDA_ALLOCATOR
        hipMalloc(addr, size);
        if (size && (*addr == 0))  {
          check_cuda_error_always("hipMemGetInfo", __FILE__, __LINE__);	
          int gpu = -1;
          hipGetDevice(&gpu);
          size_t free, total;
          hipMemGetInfo(&free, &total);
          check_cuda_error("hipMemGetInfo", __FILE__, __LINE__);	
          printf("allocating memory size %ld failed on GPU %d, total %ld, free %ld\n", size, gpu, total, free);
        }
        check_cuda_error("hipMalloc", __FILE__, __LINE__);
#else
        HeapAllocator *allocator = get_gpu_allocator();
        assert(allocator);
        addr = allocator->allocate(size);
#endif
        if (addr == nullptr) {
          size_t free, total;
          hipMemGetInfo(&free, &total);
          fprintf(stderr, "Failed to allocate memory size %f Kbytes, free memory %f Kbytes, total %f Kbytes.\n",
                  float(size)/1000., float(free)/1000., float(total)/1000.);
        }
      }
    return addr;

  }

  HeapAllocator *HeapManager::get_gpu_allocator()
  {
    int device(-1);
    hipGetDevice(&device);
    assert(device >= 0);
    assert(device <= HeapManager::max_device_ids);
    if (m_gpu_heap_allocator[device] == nullptr)
      m_gpu_heap_allocator[device] = new HeapAllocator();
    return m_gpu_heap_allocator[device];
  }

  void HeapManager::Free(Memory_Type type, void *addr)
  {
    if (type == CPU_HEAP)
      {
        free(addr);

      }
    else if (type == GPU_HEAP)
      {
#ifdef USE_CUDA_ALLOCATOR
        hipFree(addr);
#else
        get_gpu_allocator()->deallocate(addr);
#endif
      }
  }

  void *GpuHostAllocator(size_t size)
  {
    return gHeapManager.Malloc(CPU_HEAP, size);
  }

  void *GpuDeviceAllocator(size_t size)
  {
    return gHeapManager.Malloc(GPU_HEAP, size);
  }

  void GpuHostDeleter(void *ptr)
  {
    gHeapManager.Free(CPU_HEAP, ptr);
  }

  void EmptyDeviceDeleter(void *ptr, size_t size)
  {
  }

  void GpuDeviceDeleter(void *ptr, size_t size)
  {
    gHeapManager.Free(GPU_HEAP, ptr);
  }

}
